/*
 * reference: CUDA Samples deviceQuery.cpp
 * Compiling: $ nvcc gpu_info_check.cu
 * 
 */

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "./utils/book.h"

int main( void ) {


    int count;
    HANDLE_ERROR( hipGetDeviceCount( &count ) ); // get devices number
    int driverVersion = 0, runtimeVersion = 0;
    for (int i=0; i< count; i++) {

        hipSetDevice(i); // set the device to query
        hipDeviceProp_t  deviceProp;
        HANDLE_ERROR( hipGetDeviceProperties( &deviceProp, i ) );
        printf( "   --- General Information for device %d ---\n", i );
        printf("Device %d: \"%s\"\n", i, deviceProp.name);

        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);

        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

        char msg[256];
        sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);

        printf( "Compute capability:  %d.%d\n", deviceProp.major, deviceProp.minor );
        printf( "Clock rate:  %d\n", deviceProp.clockRate );
        printf( "Device copy overlap:  " );
        if (deviceProp.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (deviceProp.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", deviceProp.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", deviceProp.totalConstMem );
        printf( "Max mem pitch:  %ld\n", deviceProp.memPitch );
        printf( "32-bits register per block", deviceProp.regsPerBlock);
        printf( "Texture Alignment:  %ld\n", deviceProp.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Shared mem per mp:  %ld\n", deviceProp.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", deviceProp.regsPerBlock );
        printf( "Threads in warp:  %d\n", deviceProp.warpSize );
        printf( "Max threads per block:  %d\n",
                    deviceProp.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
                    deviceProp.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
                    deviceProp.maxGridSize[2] );
        printf( "Memory Bus Width in bits: %d\n",deviceProp.memoryBusWidth);
        printf( "\n" );
    }
}
